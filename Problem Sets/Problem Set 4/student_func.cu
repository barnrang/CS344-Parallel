#include "hip/hip_runtime.h"
//Udacity HW 4
//Radix Sorting

#include "utils.h"
#include <thrust/host_vector.h>
#include <math.h>

/* Red Eye Removal
   ===============
   
   For this assignment we are implementing red eye removal.  This is
   accomplished by first creating a score for every pixel that tells us how
   likely it is to be a red eye pixel.  We have already done this for you - you
   are receiving the scores and need to sort them in ascending order so that we
   know which pixels to alter to remove the red eye.

   Note: ascending order == smallest to largest

   Each score is associated with a position, when you sort the scores, you must
   also move the positions accordingly.

   Implementing Parallel Radix Sort with CUDA
   ==========================================

   The basic idea is to construct a histogram on each pass of how many of each
   "digit" there are.   Then we scan this histogram so that we know where to put
   the output of each digit.  For example, the first 1 must come after all the
   0s so we have to know how many 0s there are to be able to start moving 1s
   into the correct position.

   1) Histogram of the number of occurrences of each digit
   2) Exclusive Prefix Sum of Histogram
   3) Determine relative offset of each digit
        For example [0 0 1 1 0 0 1]
                ->  [0 1 0 1 2 3 2]
   4) Combine the results of steps 2 & 3 to determine the final
      output location for each element and move it there

   LSB Radix sort is an out-of-place sort and you will need to ping-pong values
   between the input and output buffers we have provided.  Make sure the final
   sorted results end up in the output buffer!  Hint: You may need to do a copy
   at the end.

 */

#ifndef max
#define max( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef min
#define min( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

const int block_size = 1024;
const int DIM = 32;
const int MAX_THREADS_PER_BLOCK = 65535;
const int FIND_MAX_THREADS = 4096; //allocate to shared memory

__global__ findMax(unsigned int* const d_inputVals,
unsigned int *d_collectMax,
const size_t numElems)
{
  __shared__ unsigned int s_inputVals[FIND_MAX_THREADS];
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < numElems) s_inputVals[idx] = d_inputVals[idx];
  else s_inputVals[idx] = 0;
  __syncthreads();

  int half = FIND_MAX_THREADS / 2;
  while (half != 0) {
    if (idx < half) {
      s_inputVals[idx] = max(s_inputVals[idx], s_inputVals[idx + half]);
    }
    half /= 2;
    __syncthreads();
  }
  d_collectMax[blockIdx.x] = s_inputVals[0];

}

__global__ scanSB(unsigned int* const d_inputVals, 
  unsigned int *d_collectScan,
  unsigned int *d_collectSumScan,
  unsigned int *d_sumBlock,
  unsigned int pos,
  size_t const numElems,
  unsigned int compare) 
{
  __shared__ unsigned int s_inputVals[FIND_MAX_THREADS];
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < numElems){
    s_inputVals[idx] = d_inputVals[idx] & pos == compare;
    d_collectScan[idx] = s_inputVals[idx];
  }
  __syncthreads();

  int dist = 1;

  while (dist < FIND_MAX_THREADS) {
    if ((idx - dist) >= 0) {
      s_inputVals[idx] += s_inputVals[idx - dist];
    }
    dist *= 2;
    __syncthreads();
  }
  d_collectSumScan[idx] = s_inputVals[threadIdx.x];
  d_sumBlock[blockIdx.x] = s_inputVals[FIND_MAX_THREADS - 1];
}

__global__ reduceBlockSum(unsigned int *d_sumBlock,
const size_t numMaxBlock)
{
  __shared__ unsigned int s_sumBlock[numMaxBlock];
  int idx = threadIdx.x;
  if(idx >= numMaxBlock) return;
  s_sumBlock[idx] = d_sumBlock[idx];
  __syncthreads();

  int dist = 1;
  while (dist < numMaxBlock) {
    if ((idx - dist) >= 0) {
      s_sumBlock[idx] += s_sumBlock[idx - dist];
    }
    dist *= 2;
    __syncthreads();
  }
  d_sumBlock[idx] = s_sumBlock[idx];
}

__global__ mergeScan(unsigned int* const d_inputVals,
unsigned int *d_collectScan,
unsigned int *d_collectSumScan,
unsigned int *d_sumBlock,
unsigned int *d_interVals,
unsigned int *d_interPos,
unsigned int offset)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (d_collectScan[idx] == 0) return;
  d_interVals[d_collectSumScan[idx] + d_sumBlock[max(blockIdx.x - 1, 0) + offset]] = d_inputVals[idx];
  d_interPos[d_collectSumScan[idx] + d_sumBlock[max(blockIdx.x - 1, 0) + offset]] = d_inputPos[idx];
}

__global__ copyData(unsigned int* const d_inputVals, 
  unsigned int *d_interVals, 
  size_t const numElems)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx >= numElems) return;
  d_inputVals[idx] = d_interVals[idx];
}

void your_sort(unsigned int* const d_inputVals,
               unsigned int* const d_inputPos,
               unsigned int* const d_outputVals,
               unsigned int* const d_outputPos,
               const size_t numElems)
{ 
  //TODO

  // P.1 search for maximum
  unsigned int *d_collectMax;
  int numMaxBlock = (numElems + FIND_MAX_THREADS - 1)/FIND_MAX_THREADS;
  checkCudaErrors(hipMallocManaged(&d_collectMax, sizeof(unsigned int) * numMaxBlock));
  findMax <<<numMaxBlock,FIND_MAX_THREADS>>>(d_inputVals, d_collectMax, numElems);
  findMax <<<1, numMaxBlock>>>(d_collectMax, d_collectMax, numMaxBlock);
  unsigned int MAX = d_collectMax[0];
  checkCudaErrors(hipFree(d_collectMax));

  // P.2 Scan and Compact
  int N = log2(MAX);
  unsigned int MSB = 1;
  unsigned int *d_collectSumScan, *d_interVals, *d_interPos, *d_sumBlock;
  bool *d_collectScan;
  checkCudaErrors(hipMallocManaged(&d_collectSumScan, sizeof(unsigned int) * numElems));
  checkCudaErrors(hipMallocManaged(&d_collectScan, sizeof(bool) * numElems));
  checkCudaErrors(hipMallocManaged(&d_interVals, sizeof(unsigned int) * numElems));
  checkCudaErrors(hipMallocManaged(&d_interPos, sizeof(unsigned int) * numElems));
  checkCudaErrors(hipMallocManaged(&d_sumBlock, sizeof(unsigned int) * numMaxBlock));
  for (int i = 0; i < N; ++i) {
    /*
    1. Predict & Scan through each block
    2. Reduce sum for each block
    3. compact elements by merging all block
    */
    scanSB<<<numMaxBlock,FIND_MAX_THREADS>>>(d_inputVals, 
      d_collectScan, d_sumBlock, MSB, numElems, 0);
    reduceBlockSum<<<1,numMaxBlock>>>(d_sumBlock, numMaxBlock);
    mergeScan<<<numMaxBlock, FIND_MAX_THREADS>>>(d_inputVals,
      d_collectScan,
      d_collectSumScan,
      d_sumBlock,
      d_interVals,
      0);
    int offset = d_sumBlock[numMaxBlock - 1];
    scanSB<<<numMaxBlock,FIND_MAX_THREADS>>>(d_inputVals, 
      d_collectScan, d_sumBlock, MSB, numElems, 1);
    reduceBlockSum<<<1,numMaxBlock>>>(d_sumBlock, numMaxBlock);
    mergeScan<<<numMaxBlock, FIND_MAX_THREADS>>>(d_inputVals,
      d_collectScan,
      d_collectSumScan,
      d_sumBlock,
      d_interVals,
      d_interPos,
      offset);
      
    checkCudaErrors(hipMemcpy(d_inputPos, d_interPos, sizeof(unsigned int) * numElems, hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy(d_inputVals, d_interVals, sizeof(unsigned int) * numElems, hipMemcpyDeviceToDevice));
    MSB *= 2;
  }

  checkCudaErrors(hipMemcpy(d_outputPos, d_inputPos, sizeof(unsigned int) * numElems, hipMemcpyDeviceToDevice));
  checkCudaErrors(hipMemcpy(d_outputVals, d_inputVals, sizeof(unsigned int) * numElems, hipMemcpyDeviceToDevice));
  //PUT YOUR SORT HERE
  checkCudaErrors(hipFree(d_collectSumScan));
  checkCudaErrors(hipFree(d_collectScan));
  checkCudaErrors(hipFree(d_interVals));
  checkCudaErrors(hipFree(d_interPos));
  checkCudaErrors(hipFree(d_sumBlock));
}
