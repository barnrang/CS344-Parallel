#include "hip/hip_runtime.h"
//Udacity HW 6
//Poisson Blending

/* Background
   ==========

   The goal for this assignment is to take one image (the source) and
   paste it into another image (the destination) attempting to match the
   two images so that the pasting is non-obvious. This is
   known as a "seamless clone".

   The basic ideas are as follows:

   1) Figure out the interior and border of the source image
   2) Use the values of the border pixels in the destination image 
      as boundary conditions for solving a Poisson equation that tells
      us how to blend the images.
   
      No pixels from the destination except pixels on the border
      are used to compute the match.

   Solving the Poisson Equation
   ============================

   There are multiple ways to solve this equation - we choose an iterative
   method - specifically the Jacobi method. Iterative methods start with
   a guess of the solution and then iterate to try and improve the guess
   until it stops changing.  If the problem was well-suited for the method
   then it will stop and where it stops will be the solution.

   The Jacobi method is the simplest iterative method and converges slowly - 
   that is we need a lot of iterations to get to the answer, but it is the
   easiest method to write.

   Jacobi Iterations
   =================

   Our initial guess is going to be the source image itself.  This is a pretty
   good guess for what the blended image will look like and it means that
   we won't have to do as many iterations compared to if we had started far
   from the final solution.

   ImageGuess_prev (Floating point)
   ImageGuess_next (Floating point)

   DestinationImg
   SourceImg

   Follow these steps to implement one iteration:

   1) For every pixel p in the interior, compute two sums over the four neighboring pixels:
      Sum1: If the neighbor is in the interior then += ImageGuess_prev[neighbor]
             else if the neighbor in on the border then += DestinationImg[neighbor]

      Sum2: += SourceImg[p] - SourceImg[neighbor]   (for all four neighbors)

   2) Calculate the new pixel value:
      float newVal= (Sum1 + Sum2) / 4.f  <------ Notice that the result is FLOATING POINT
      ImageGuess_next[p] = min(255, max(0, newVal)); //clamp to [0, 255]


    In this assignment we will do 800 iterations.
   */


/*
Channel Separate from P.2
*/

#include "utils.h"
#include <thrust/host_vector.h>
#include <vector>

#ifndef max
#define max( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef min
#define min( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

__global__
void separateChannels(const uchar4* const inputImageRGBA,
                      int numRows,
                      int numCols,
                      unsigned char* const redChannel,
                      unsigned char* const greenChannel,
                      unsigned char* const blueChannel)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if ( x < numRows && y < numCols) {
    int offset = x * numCols + y;
    uchar4 rgba = inputImageRGBA[offset];
    redChannel[offset] = (unsigned char)rgba.x;
    greenChannel[offset] = (unsigned char)rgba.y;
    blueChannel[offset] = (unsigned char)rgba.z;
  }
}

__global__ 
void sourceMask(const uchar4* const sourceImg,
                      unsigned char* sourceMask,
                      const size_t numRowsSource, 
                      const size_t numColsSource)
{
  int idx = threadIdx.x, idy = threadIdx.y, bdx = blockIdx.x, bdy = blockIdx.y;
  int dimx = blockDim.x, dimy = blockDim.y;
  int x = bdx * dimx + idx;
  int y = bdy * dimy + idy;
  int offset = x * numColsSource + y;
  if (x < numRowsSource && y < numColsSource){
    sourceMask[offset] = ((sourceImg[offset].x + sourceImg[offset].y 
      + sourceImg[offset].z) < 255 * 3) ? 1 : 0;
  }
}

__global__ 
void isStrictInterior(
  unsigned char* sourceMask,
  //unsigned char* strictInteriorPixels,
  unsigned char* borderPixels,
  unsigned char* interiorPixels,
  const size_t numRowsSource, 
  const size_t numColsSource
)
{
  int idx = threadIdx.x, idy = threadIdx.y, bdx = blockIdx.x, bdy = blockIdx.y;
  int dimx = blockDim.x, dimy = blockDim.y;
  int x = bdx * dimx + idx;
  int y = bdy * dimy + idy;
  if (x >= numRowsSource || y >= numColsSource) return;
  int offset = x * numColsSource + y;
  //if (offset >= numColsSource * numRowsSource) return;
  if (!sourceMask[offset]) {
    borderPixels[offset] = 0;
    interiorPixels[offset] = 0;
    //strictInteriorPixels[offset] = 0;
  }
  else if (sourceMask[(x - 1) * numColsSource + y] && sourceMask[(x + 1) * numColsSource + y]
    && sourceMask[x * numColsSource + y - 1] && sourceMask[x * numColsSource + y + 1]){
      //strictInteriorPixels[offset] = 1;
      interiorPixels[offset] = 1;
      borderPixels[offset] = 0;
    }
  else {
    //strictInteriorPixels[offset] = 0;
    interiorPixels[offset] = 0;
    borderPixels[offset] = 1;
  }
}

__global__ 
void debugMask(
  unsigned char* sourceMask,
  uchar4* d_out,
  const size_t numRowsSource, 
  const size_t numColsSource
)
{
  int idx = threadIdx.x, idy = threadIdx.y, bdx = blockIdx.x, bdy = blockIdx.y;
  int dimx = blockDim.x, dimy = blockDim.y;
  int x = bdx * dimx + idx;
  int y = bdy * dimy + idy;
  if (x >= numRowsSource || y >= numColsSource) return;
  int offset = x * numColsSource + y;

  //if (offset >= (numRowsSource * numColsSource)) return;
  if (sourceMask[offset]){
    d_out[offset].x = 255;
    d_out[offset].y = 255;
    d_out[offset].z = 255;
    d_out[offset].w= 255;
  }
  else {
    d_out[offset].x = 0;
    d_out[offset].y = 0;
    d_out[offset].z = 0;
    d_out[offset].w= 255;
  }
}

__global__ 
void debugBorder(
  unsigned char* sourceMask,
  unsigned char* borderPixels,
  unsigned char* interiorPixels,
  uchar4* d_out,
  const size_t numRowsSource, 
  const size_t numColsSource
)
{
  int idx = threadIdx.x, idy = threadIdx.y, bdx = blockIdx.x, bdy = blockIdx.y;
  int dimx = blockDim.x, dimy = blockDim.y;
  int x = bdx * dimx + idx;
  int y = bdy * dimy + idy;
  if (x >= numRowsSource || y >= numColsSource) return;
  int offset = x * numColsSource + y;

  //if (offset >= (numRowsSource * numColsSource)) return;
  if (borderPixels[offset]){
    d_out[offset].x = 220;
    d_out[offset].y = 20;
    d_out[offset].z = 60;
    d_out[offset].w= 255;
  }
  else if (interiorPixels[offset]){
    d_out[offset].x = 0;
    d_out[offset].y = 128;
    d_out[offset].z = 0;
    d_out[offset].w= 255;
  }
  else {
    d_out[offset].x = 0;
    d_out[offset].y = 0;
    d_out[offset].z = 0;
    d_out[offset].w= 255;
  }
}

__global__
void copy(
  float* to_this,
  unsigned char* from,
  const size_t numRowsSource, 
  const size_t numColsSource
)
{
  int idx = threadIdx.x, idy = threadIdx.y, bdx = blockIdx.x, bdy = blockIdx.y;
  int dimx = blockDim.x, dimy = blockDim.y;
  int x = bdx * dimx + idx;
  int y = bdy * dimy + idy;
  int offset = x * numColsSource + y;
  if (x >= numRowsSource || y >= numColsSource) return;
  //if (offset >= numColsSource * numRowsSource) return;
  to_this[offset] = (float)from[offset];

}

__global__
void computeG(
  unsigned char* channel,
  float* g,
  unsigned char* d_interiorPixels,
  const size_t numRowsSource, 
  const size_t numColsSource
)
{
  int idx = threadIdx.x, idy = threadIdx.y, bdx = blockIdx.x, bdy = blockIdx.y;
  int dimx = blockDim.x, dimy = blockDim.y;
  int x = bdx * dimx + idx;
  int y = bdy * dimy + idy;
  if (x >= numRowsSource || y >= numColsSource) return;
  int offset = x * numColsSource + y;
  //if (offset >= numRowsSource * numColsSource) return;
  if (d_interiorPixels[offset]) {
    float sum = 4.f * channel[offset];
    sum -= (float)channel[offset - numColsSource] + (float)channel[offset + numColsSource];
    sum -= (float)channel[offset - 1] + (float)channel[offset + 1];
    g[offset] = sum;
  } else {
    g[offset] = 0;
  }
}

__global__
void naiveJacobi(
  const unsigned char* const d_destImg,
  const unsigned char* const d_interiorPixels,
  const unsigned char* const d_borderPixels,
  const size_t numColsSource,
  const size_t numRowsSource,
  float* const f,
  const float* const g,
  float* const f_next
)
{
  int idx = threadIdx.x, idy = threadIdx.y, bdx = blockIdx.x, bdy = blockIdx.y;
  int dimx = blockDim.x, dimy = blockDim.y;
  int x = bdx * dimx + idx;
  int y = bdy * dimy + idy;
  if (x >= numRowsSource || y >= numColsSource) return;
  int offset = x * numColsSource + y;
  float sum = 0.f;
  int iterate[4] = {offset - 1, offset + 1, (int)(offset - numColsSource), (int)(offset + numColsSource)};
  if (!d_interiorPixels[offset]) return;
  for (int i = 0; i < 4; i++) {
    int coor = iterate[i];
    sum += d_interiorPixels[coor] * f[coor] + (1 - d_interiorPixels[coor]) * d_destImg[coor];
  }
  float f_next_val = (sum + g[offset]) / 4.f;
  f_next_val = min(255.f, max(0.f, f_next_val));
  f_next[offset] = f_next_val;
}

__global__
void pasteImage(
  uchar4* d_destImg,
  const unsigned char* const d_interiorPixels,
  float* blendedValsRed,
  float* blendedValsGreen,
  float* blendedValsBlue,
  const size_t numColsSource,
  const size_t numRowsSource
)
{
  int idx = threadIdx.x, idy = threadIdx.y, bdx = blockIdx.x, bdy = blockIdx.y;
  int dimx = blockDim.x, dimy = blockDim.y;
  int x = bdx * dimx + idx;
  int y = bdy * dimy + idy;
  if (x >= numRowsSource || y >= numColsSource) return;
  int offset = x * numColsSource + y;
  //if (offset >= numColsSource * numRowsSource) return;
  if (!d_interiorPixels[offset]) return;

  d_destImg[offset].x = (unsigned char)blendedValsRed[offset];
  d_destImg[offset].y = (unsigned char)blendedValsGreen[offset];
  d_destImg[offset].z = (unsigned char)blendedValsBlue[offset];
  
}

void your_blend(const uchar4* const h_sourceImg,  //IN
                const size_t numRowsSource, const size_t numColsSource,
                const uchar4* const h_destImg, //IN
                uchar4* const h_blendedImg) //OUT
{

  const unsigned int KERNEL_DIM = 16;
  unsigned int numPixel = numColsSource * numRowsSource;
  unsigned char* d_sourceMask;
  //unsigned char* d_strictInteriorPixels;
  unsigned char* d_borderPixels;
  unsigned char* d_interiorPixels;
  uchar4* d_sourceImg;
  uchar4* d_destImg;
  uchar4* d_blendedImg;
  checkCudaErrors(hipMallocManaged(&d_sourceImg, sizeof(uchar4) * numPixel));
  checkCudaErrors(hipMemcpy(d_sourceImg, h_sourceImg, sizeof(uchar4) * numPixel, 
    hipMemcpyHostToDevice));
  
  checkCudaErrors(hipMallocManaged(&d_destImg, sizeof(uchar4) * numPixel));
  checkCudaErrors(hipMemcpy(d_destImg, h_destImg, sizeof(uchar4) * numPixel, 
    hipMemcpyHostToDevice));
  
  checkCudaErrors(hipMallocManaged(&d_blendedImg, sizeof(uchar4) * numPixel));
  checkCudaErrors(hipMallocManaged(&d_sourceMask, sizeof(unsigned char) * numPixel));
  //checkCudaErrors(hipMallocManaged(&d_strictInteriorPixels, sizeof(unsigned char) * numPixel));
  checkCudaErrors(hipMallocManaged(&d_borderPixels, sizeof(unsigned char) * numPixel));
  checkCudaErrors(hipMallocManaged(&d_interiorPixels, sizeof(unsigned char) * numPixel));


  //Step 1
  int rowBlock = (numRowsSource + KERNEL_DIM - 1)/KERNEL_DIM; 
  int colBlock = (numColsSource + KERNEL_DIM - 1)/KERNEL_DIM; 
  const dim3 blockSize(rowBlock, colBlock, 1);
  const dim3 kernelSize(KERNEL_DIM, KERNEL_DIM, 1);
  sourceMask<<<blockSize, kernelSize>>>(
    d_sourceImg,
    d_sourceMask,
    numRowsSource, 
    numColsSource
  );
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

  // debugMask<<<blockSize, kernelSize>>>(
  //   d_sourceMask,
  //   d_blendedImg,
  //   numRowsSource, 
  //   numColsSource
  // );
  // hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
  
  // checkCudaErrors(hipMemcpy(h_blendedImg, d_blendedImg, sizeof(uchar4) * numPixel, 
  //   hipMemcpyDeviceToHost));
  

  //Step 2

  isStrictInterior<<<blockSize, kernelSize>>>(
    d_sourceMask,
    d_borderPixels,
    d_interiorPixels,
    numRowsSource, 
    numColsSource
  );
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

  debugBorder<<<blockSize, kernelSize>>>(
    d_sourceMask,
    d_borderPixels,
    d_interiorPixels,
    d_blendedImg,
    numRowsSource, 
    numColsSource
  );
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

  checkCudaErrors(hipMemcpy(h_blendedImg, d_blendedImg, sizeof(uchar4) * numPixel, 
    hipMemcpyDeviceToHost));

  //return;

  // Step 3

  unsigned char* d_srcRedChannel;
  unsigned char* d_srcGreenChannel;
  unsigned char* d_srcBlueChannel;

  unsigned char* d_destRedChannel;
  unsigned char* d_destGreenChannel;
  unsigned char* d_destBlueChannel;

  checkCudaErrors(hipMallocManaged(&d_srcRedChannel, sizeof(unsigned char) * numPixel));
  checkCudaErrors(hipMallocManaged(&d_srcGreenChannel, sizeof(unsigned char) * numPixel));
  checkCudaErrors(hipMallocManaged(&d_srcBlueChannel, sizeof(unsigned char) * numPixel));
  checkCudaErrors(hipMallocManaged(&d_destRedChannel, sizeof(unsigned char) * numPixel));
  checkCudaErrors(hipMallocManaged(&d_destGreenChannel, sizeof(unsigned char) * numPixel));
  checkCudaErrors(hipMallocManaged(&d_destBlueChannel, sizeof(unsigned char) * numPixel));

  separateChannels<<<blockSize, kernelSize>>>(
    d_sourceImg,
    numRowsSource,
    numColsSource,
    d_srcRedChannel,
    d_srcGreenChannel,
    d_srcBlueChannel
  );

  separateChannels<<<blockSize, kernelSize>>>(
    d_destImg,
    numRowsSource,
    numColsSource,
    d_destRedChannel,
    d_destGreenChannel,
    d_destBlueChannel
  );
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

  // Step 4

  float *blendedValsRed_1;
  float *blendedValsRed_2;
  float *blendedValsGreen_1;
  float *blendedValsGreen_2;
  float *blendedValsBlue_1;
  float *blendedValsBlue_2;
  float *g_red;
  float *g_green;
  float *g_blue;

  checkCudaErrors(hipMallocManaged(&blendedValsRed_1, sizeof(float) * numPixel));
  checkCudaErrors(hipMallocManaged(&blendedValsRed_2, sizeof(float) * numPixel));
  checkCudaErrors(hipMallocManaged(&blendedValsGreen_1, sizeof(float) * numPixel));
  checkCudaErrors(hipMallocManaged(&blendedValsGreen_2, sizeof(float) * numPixel));
  checkCudaErrors(hipMallocManaged(&blendedValsBlue_1, sizeof(float) * numPixel));
  checkCudaErrors(hipMallocManaged(&blendedValsBlue_2, sizeof(float) * numPixel));
  checkCudaErrors(hipMallocManaged(&g_blue, sizeof(float) * numPixel));
  checkCudaErrors(hipMallocManaged(&g_green, sizeof(float) * numPixel));
  checkCudaErrors(hipMallocManaged(&g_red, sizeof(float) * numPixel));

  copy<<<blockSize, kernelSize>>>(
    blendedValsRed_1, d_srcRedChannel, numRowsSource, numColsSource
  );
  copy<<<blockSize, kernelSize>>>(
    blendedValsRed_2, d_srcRedChannel, numRowsSource, numColsSource
  );
  copy<<<blockSize, kernelSize>>>(
    blendedValsGreen_1, d_srcGreenChannel, numRowsSource, numColsSource
  );
  copy<<<blockSize, kernelSize>>>(
    blendedValsGreen_2, d_srcGreenChannel, numRowsSource, numColsSource
  );
  copy<<<blockSize, kernelSize>>>(
    blendedValsBlue_1, d_srcBlueChannel, numRowsSource, numColsSource
  );
  copy<<<blockSize, kernelSize>>>(
    blendedValsBlue_2, d_srcBlueChannel, numRowsSource, numColsSource
  );
  

  // checkCudaErrors(hipMemcpy(blendedValsRed_1, d_srcRedChannel, sizeof(float) * numPixel,
  //   hipMemcpyDeviceToDevice));
  // checkCudaErrors(hipMemcpy(blendedValsRed_2, d_srcRedChannel, sizeof(float) * numPixel,
  //   hipMemcpyDeviceToDevice));
  // checkCudaErrors(hipMemcpy(blendedValsGreen_1, d_srcGreenChannel, sizeof(float) * numPixel,
  //   hipMemcpyDeviceToDevice));
  // checkCudaErrors(hipMemcpy(blendedValsGreen_2, d_srcGreenChannel, sizeof(float) * numPixel,
  //   hipMemcpyDeviceToDevice));
  // checkCudaErrors(hipMemcpy(blendedValsBlue_1, d_srcBlueChannel, sizeof(float) * numPixel,
  //   hipMemcpyDeviceToDevice));
  // checkCudaErrors(hipMemcpy(blendedValsBlue_2, d_srcBlueChannel, sizeof(float) * numPixel,
  //   hipMemcpyDeviceToDevice));

  computeG<<<blockSize, kernelSize>>>( d_srcRedChannel, g_red, 
    d_interiorPixels, numRowsSource, numColsSource);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
  computeG<<<blockSize, kernelSize>>>( d_srcBlueChannel, g_blue, 
    d_interiorPixels, numRowsSource, numColsSource);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
  computeG<<<blockSize, kernelSize>>>( d_srcGreenChannel, g_green, 
    d_interiorPixels, numRowsSource, numColsSource);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

  // Step 5

  for (int i = 0; i < 800; ++i) {
    naiveJacobi <<<blockSize, kernelSize>>>(
      d_destRedChannel,
      d_interiorPixels,
      d_borderPixels,
      numColsSource,
      numRowsSource,
      blendedValsRed_1,
      g_red,
      blendedValsRed_2
    );

    naiveJacobi <<<blockSize, kernelSize>>>(
      d_destGreenChannel,
      d_interiorPixels,
      d_borderPixels,
      numColsSource,
      numRowsSource,
      blendedValsGreen_1,
      g_green,
      blendedValsGreen_2
    );

    naiveJacobi <<<blockSize, kernelSize>>>(
      d_destBlueChannel,
      d_interiorPixels,
      d_borderPixels,
      numColsSource,
      numRowsSource,
      blendedValsBlue_1,
      g_blue,
      blendedValsBlue_2
    );
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipMemcpy(blendedValsRed_1, blendedValsRed_2, sizeof(float) * numPixel,
    hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy(blendedValsGreen_1, blendedValsGreen_2, sizeof(float) * numPixel,
    hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy(blendedValsBlue_1, blendedValsBlue_2, sizeof(float) * numPixel,
    hipMemcpyDeviceToDevice));
    
  }

  pasteImage<<<blockSize, kernelSize>>>(
    d_destImg,
    d_interiorPixels,
    blendedValsRed_1,
    blendedValsGreen_1,
    blendedValsBlue_1,
    numColsSource,
    numRowsSource
  );

  checkCudaErrors(hipMemcpy(h_blendedImg, d_destImg, sizeof(uchar4) * numPixel,
  hipMemcpyDeviceToHost));
  

  //Free memory
  checkCudaErrors(hipFree(d_sourceImg));
  checkCudaErrors(hipFree(d_destImg));
  checkCudaErrors(hipFree(d_sourceMask));
  checkCudaErrors(hipFree(d_blendedImg));
  checkCudaErrors(hipFree(d_borderPixels));
  checkCudaErrors(hipFree(d_interiorPixels));

  checkCudaErrors(hipFree(blendedValsRed_1));
  checkCudaErrors(hipFree(blendedValsRed_2));
  checkCudaErrors(hipFree(blendedValsGreen_1));
  checkCudaErrors(hipFree(blendedValsGreen_2));
  checkCudaErrors(hipFree(blendedValsBlue_1));
  checkCudaErrors(hipFree(blendedValsBlue_2));

  checkCudaErrors(hipFree(g_blue));
  checkCudaErrors(hipFree(g_green));
  checkCudaErrors(hipFree(g_red));
  
  checkCudaErrors(hipFree(d_srcRedChannel));
  checkCudaErrors(hipFree(d_srcGreenChannel));
  checkCudaErrors(hipFree(d_srcBlueChannel));

  checkCudaErrors(hipFree(d_destRedChannel));
  checkCudaErrors(hipFree(d_destGreenChannel));
  checkCudaErrors(hipFree(d_destBlueChannel));

  /* To Recap here are the steps you need to implement
  
     1) Compute a mask of the pixels from the source image to be copied
        The pixels that shouldn't be copied are completely white, they
        have R=255, G=255, B=255.  Any other pixels SHOULD be copied.

     2) Compute the interior and border regions of the mask.  An interior
        pixel has all 4 neighbors also inside the mask.  A border pixel is
        in the mask itself, but has at least one neighbor that isn't.

     3) Separate out the incoming image into three separate channels

     4) Create two float(!) buffers for each color channel that will
        act as our guesses.  Initialize them to the respective color
        channel of the source image since that will act as our intial guess.

     5) For each color channel perform the Jacobi iteration described 
        above 800 times.

     6) Create the output image by replacing all the interior pixels
        in the destination image with the result of the Jacobi iterations.
        Just cast the floating point values to unsigned chars since we have
        already made sure to clamp them to the correct range.

      Since this is final assignment we provide little boilerplate code to
      help you.  Notice that all the input/output pointers are HOST pointers.

      You will have to allocate all of your own GPU memory and perform your own
      memcopies to get data in and out of the GPU memory.

      Remember to wrap all of your calls with checkCudaErrors() to catch any
      thing that might go wrong.  After each kernel call do:

      hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

      to catch any errors that happened while executing the kernel.
  */
}
